#include <bcl/bcl.hpp>
#include <bcl/backends/experimental/nvshmem/backend.hpp>
#include <bcl/containers/experimental/cuda/DuplQueue.hpp>
#include <bcl/containers/experimental/cuda/launch_kernel.cuh>

#include <chrono>

#define NUM_INSERTS 2*8*1024

int main(int argc, char** argv) {
  BCL::init(16);

  printf("Hello, world! I am rank %lu/%lu\n",
         BCL::rank(), BCL::nprocs());

  BCL::cuda::init(8*1024);

  size_t num_inserts = NUM_INSERTS;
  size_t insert_size = 1024;

  BCL::cuda::DuplQueue<int> queue(0, num_inserts*insert_size);

  BCL::cuda::device_vector<int, BCL::cuda::bcl_allocator<int>> values(insert_size);
  // BCL::cuda::device_vector<int> values(insert_size);
  std::vector<int> values_local(insert_size, BCL::rank());
  values.assign(values_local.begin(), values_local.end());

  BCL::cuda::barrier();
  auto begin = std::chrono::high_resolution_clock::now();

  BCL::cuda::global_launch(num_inserts,
                     [] __device__ (size_t idx, BCL::cuda::DuplQueue<int>& queue,
                                    BCL::cuda::device_vector<int, BCL::cuda::bcl_allocator<int>>& values) {
                                    // BCL::cuda::device_vector<int>& values) {
                       bool success = queue.push(values.data(), values.size());
                       if (!success) {
                         printf("AGH! I have failed!\n");
                       }
                     }, queue, values);

  hipDeviceSynchronize();
  BCL::cuda::barrier();
  auto end = std::chrono::high_resolution_clock::now();

  double duration = std::chrono::duration<double>(end - begin).count();

  double data_moved = num_inserts*insert_size*sizeof(int);

  double bw = data_moved / duration;
  double bw_gb = bw*1e-9;

  BCL::print("Total %lf s (%lf GB/s)\n", duration, bw_gb);

  if (BCL::rank() == 0) {
    BCL::cuda::launch(num_inserts,
                      [] __device__ (size_t idx, BCL::cuda::DuplQueue<int>& queue) {
                        int value = 12;
                        bool success = queue.local_pop(value);
                        // printf("%lu: %d (%s)\n", idx, value, (success) ? "success" : "failure");
                      }, queue);
    hipDeviceSynchronize();
  }
  BCL::cuda::barrier();

  BCL::print("Here...\n");

  BCL::cuda::barrier();
  BCL::print("After barrier...\n");

  BCL::finalize();
  return 0;
}
